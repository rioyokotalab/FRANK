#include "hip/hip_runtime.h"
#include "hicma/node_proxy.h"
#include "hicma/low_rank.h"
#include "hicma/gpu_batch/batch.h"
#include "hicma/util/timer.h"

#include <hipblas.h>
#include <magma_v2.h>
#include "kblas.h"
#include "testing_helper.h"
#include "batch_rand.h"
#include "batch_ara.h"

#include <iostream>
#include <fstream>

namespace hicma {

  std::vector<Dense> vecA;
  std::vector<Dense> vecB;
  std::vector<Dense*> vecC;
  std::vector<NodeProxy*> vecLR;

  void rsvd_push(NodeProxy& A, Dense& Aij, int rank) {
    vecA.push_back(Aij);
    vecLR.push_back(&A);
  }

  void gemm_push(const Dense& A, const Dense& B, Dense* C) {
#if 0
    C->gemm(A, B, CblasNoTrans, CblasNoTrans, 1, 1);
#else
    vecA.push_back(A);
    vecB.push_back(B);
    vecC.push_back(C);
#endif
  }

  void rsvd_batch() {
    int batchCount = vecA.size();
    if (batchCount == 0) return;
    double tol = 1e-7;
    int block_size = 32;
    int ara_r = 10;
    int max_m = 0;
    int max_n = 0;
    std::vector<int> h_m(batchCount);
    std::vector<int> h_n(batchCount);
    std::vector<int> h_ldm(batchCount);
    std::vector<int> h_ldn(batchCount);
    for (int b=0; b<batchCount; b++) {
      Dense A = vecA[b];
      h_m[b] = A.dim[0];
      h_n[b] = A.dim[1];
      h_ldm[b] = std::max(h_m[b],32);
      h_ldn[b] = std::max(h_n[b],32);
      max_m = std::max(max_m, h_ldm[b]);
      max_n = std::max(max_n, h_ldn[b]);
    }
    start("Allocate host");
    std::vector<double> h_A(max_m * max_n * batchCount);
    std::vector<double> h_U(max_m * max_n * batchCount);
    std::vector<double> h_V(max_n * max_n * batchCount);
    stop("Allocate host");
    start("Copy matrix");
    for (int b=0; b<batchCount; b++) {
      Dense A = vecA[b];
      for (int i=0; i<A.dim[0]; i++) {
        for (int j=0; j<A.dim[1]; j++) {
          h_A[i+j*h_ldm[b]+b*max_m*max_n] = A(i,j);
        }
      }
    }
    stop("Copy matrix");
#if 0
    start("Write matrix");
    std::ofstream file("matrix.txt");
    file << batchCount << std::endl;
    for (int b=0; b<batchCount; b++) {
      Dense A = vecA[b];
      file << A.dim[0] << std::endl;
      file << A.dim[1] << std::endl;
      for (int i=0; i<A.dim[0]; i++) {
        for (int j=0; j<A.dim[1]; j++) {
          file << A(i,j) << std::endl;
        }
      }
    }
    stop("Write matrix");
#endif
    start("Init KBLAS");
    kblasHandle_t handle;
    kblasRandState_t rand_state;
    kblasCreate(&handle);
    kblasInitRandState(handle, &rand_state, 16384*2, 0);
    kblasEnableMagma(handle);
    magma_init();
    stop("Init KBLAS");
    start("Allocate memory");
    int *d_m, *d_n, *d_k, *d_ldm, *d_ldn;
    hipMalloc( (void**)&d_m, batchCount * sizeof(int) );
    hipMalloc( (void**)&d_n, batchCount * sizeof(int) );
    hipMalloc( (void**)&d_k, batchCount * sizeof(int) );
    hipMalloc( (void**)&d_ldm, batchCount * sizeof(int) );
    hipMalloc( (void**)&d_ldn, batchCount * sizeof(int) );
    double *d_A, *d_U, *d_V;
    hipMalloc( (void**)&d_A, h_A.size() * sizeof(double) );
    hipMalloc( (void**)&d_U, h_U.size() * sizeof(double) );
    hipMalloc( (void**)&d_V, h_V.size() * sizeof(double) );
    double **p_A, **p_U, **p_V;
    hipMalloc( (void**)&p_A, batchCount * sizeof(double*) );
    hipMalloc( (void**)&p_U, batchCount * sizeof(double*) );
    hipMalloc( (void**)&p_V, batchCount * sizeof(double*) );
    stop("Allocate memory");
    start("Array of pointers");
    generateDArrayOfPointers(d_A, p_A, max_m * max_n, batchCount, 0);
    generateDArrayOfPointers(d_U, p_U, max_m * max_n, batchCount, 0);
    generateDArrayOfPointers(d_V, p_V, max_n * max_n, batchCount, 0);
    stop("Array of pointers");
    start("Copy to device");
    kblas_ara_batch_wsquery<double>(handle, block_size, batchCount);
    kblasAllocateWorkspace(handle);
    hipMemcpy(d_m, &h_m[0], batchCount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_n, &h_n[0], batchCount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ldm, &h_ldm[0], batchCount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ldn, &h_ldn[0], batchCount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A, &h_A[0], h_A.size() * sizeof(double), hipMemcpyHostToDevice);
    stop("Copy to device");
    start("Batched SVD");
    kblas_ara_batched(handle, d_m, d_n, p_A, d_ldm, p_U, d_ldm, p_V, d_ldn, d_k,
                      tol, max_m, max_n, max_n, block_size, ara_r, rand_state, batchCount);
    hipDeviceSynchronize();
    stop("Batched SVD");
    start("Copy to host");
    std::vector<int> h_k(batchCount);
    hipMemcpy(&h_k[0], d_k, batchCount * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_U[0], d_U, h_U.size() * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(&h_V[0], d_V, h_V.size() * sizeof(double), hipMemcpyDeviceToHost);
    stop("Copy to host");
    start("Copy to LR");
    for(int b=0; b<batchCount; b++) {
      assert(h_k[b] != 0);
      LowRank LR(vecA[b].dim[0], vecA[b].dim[1], h_k[b]);
      //std::cout << LR.dim[0] << " " << LR.dim[1] << " " << LR.rank << std::endl;
      Dense A = vecA[b];
      for (int i=0; i<LR.dim[0]; i++) {
        for (int j=0; j<LR.rank; j++) {
          LR.U(i,j) = h_U[i+j*h_ldm[b]+b*max_m*max_n];
        }
      }
      for (int i=0; i<LR.rank; i++) {
        for (int j=0; j<LR.dim[1]; j++) {
          LR.V(i,j) = h_V[i*h_ldn[b]+j+b*max_n*max_n];
        }
        LR.S(i,i) = 1;
      }
      *vecLR[b] = LR;
    }
    stop("Copy to LR");
    start("Free memory");
    vecA.clear();
    vecLR.clear();
    hipFree(p_A);
    hipFree(p_U);
    hipFree(p_V);
    hipFree(d_A);
    hipFree(d_U);
    hipFree(d_V);
    hipFree(d_m);
    hipFree(d_n);
    hipFree(d_k);
    hipFree(d_ldm);
    hipFree(d_ldn);
    kblasDestroy(&handle);
    stop("Free memory");
  }

  void gemm_batch() {
    int batchCount = vecA.size();
    if (batchCount == 0) return;
    double alpha = 1;
    double beta = 1;
    std::vector<int> h_m(batchCount);
    std::vector<int> h_n(batchCount);
    std::vector<int> h_k(batchCount);
    int max_m = 0, max_n = 0, max_k = 0;
    for(int b=0; b<batchCount; b++){
      Dense A = vecA[b];
      Dense B = vecB[b];
      h_m[b] = A.dim[0];
      h_n[b] = B.dim[1];
      h_k[b] = A.dim[1];
      max_m = std::max(max_m,h_m[b]);
      max_n = std::max(max_n,h_n[b]);
      max_k = std::max(max_k,h_k[b]);
    }
    start("Allocate host");
    std::vector<double> h_A(max_m * max_k * batchCount);
    std::vector<double> h_B(max_k * max_n * batchCount);
    std::vector<double> h_C(max_m * max_n * batchCount);
    stop("Allocate host");
    start("Copy matrix");
    for (int b=0; b<batchCount; b++) {
      Dense A = vecA[b];
      for (int i=0; i<A.dim[0]; i++) {
        for (int j=0; j<A.dim[1]; j++) {
          h_A[i+j*A.dim[0]+b*max_m*max_k] = A(i,j);
        }
      }
      Dense B = vecB[b];
      for (int i=0; i<B.dim[0]; i++) {
        for (int j=0; j<B.dim[1]; j++) {
          h_B[i+j*B.dim[0]+b*max_k*max_n] = B(i,j);
        }
      }
      Dense C = *vecC[b];
      for (int i=0; i<C.dim[0]; i++) {
        for (int j=0; j<C.dim[1]; j++) {
          h_C[i+j*C.dim[0]+b*max_m*max_n] = C(i,j);
        }
      }
    }
    stop("Copy matrix");
    start("Init KBLAS");
    kblasHandle_t handle;
    kblasCreate(&handle);
    kblasEnableMagma(handle);
    magma_init();
    stop("Init KBLAS");
    start("Allocate memory");
    int *d_m, *d_n, *d_k;
    hipMalloc( (void**)&d_m, batchCount * sizeof(int) );
    hipMalloc( (void**)&d_n, batchCount * sizeof(int) );
    hipMalloc( (void**)&d_k, batchCount * sizeof(int) );
    double *d_A, *d_B, *d_C;
    hipMalloc( (void**)&d_A, h_A.size() * sizeof(double) );
    hipMalloc( (void**)&d_B, h_B.size() * sizeof(double) );
    hipMalloc( (void**)&d_C, h_C.size() * sizeof(double) );
    double **p_A, **p_B, **p_C;
    hipMalloc( (void**)&p_A, batchCount * sizeof(double*) );
    hipMalloc( (void**)&p_B, batchCount * sizeof(double*) );
    hipMalloc( (void**)&p_C, batchCount * sizeof(double*) );
    stop("Allocate memory");
    start("Array of pointers");
    generateDArrayOfPointers(d_A, p_A, max_m * max_k, batchCount, 0);
    generateDArrayOfPointers(d_B, p_B, max_k * max_n, batchCount, 0);
    generateDArrayOfPointers(d_C, p_C, max_m * max_n, batchCount, 0);
    stop("Array of pointers");
    start("Copy to device");
    kblas_gemm_batch_nonuniform_wsquery(handle);
    kblasAllocateWorkspace(handle);
    hipMemcpy(d_m, &h_m[0], batchCount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_n, &h_n[0], batchCount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_k, &h_k[0], batchCount * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A, &h_A[0], h_A.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, &h_B[0], h_B.size() * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_C, &h_C[0], h_C.size() * sizeof(double), hipMemcpyHostToDevice);
    stop("Copy to device");
    start("Batched GEMM");
    kblas_gemm_batch(handle, 'N', 'N', d_m, d_n, d_k, max_m, max_n, max_k,
                     alpha, (const double**)p_A, d_m, (const double**)p_B, d_k,
                     beta, p_C, d_m, batchCount );
    stop("Batched GEMM");
    start("Copy to host");
    hipMemcpy(&h_C[0], d_C, max_m * max_n * batchCount * sizeof(double), hipMemcpyDeviceToHost);
    stop("Copy to host");
    start("Copy to C");
    for (int b=0; b<batchCount; b++) {
      Dense* C = vecC[b];
#if 0
      Dense A = vecA[b];
      Dense B = vecB[b];
      C->gemm(A, B, CblasNoTrans, CblasNoTrans, 1, 1);
#else
      for (int i=0; i<C->dim[0]; i++) {
        for (int j=0; j<C->dim[1]; j++) {
          (*C)(i,j) += h_C[i+j*C->dim[0]+b*max_m*max_n];
        }
      }
#endif
    }
    stop("Copy to C");
    start("Free memory");
    vecA.clear();
    vecB.clear();
    vecC.clear();
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_m);
    hipFree(d_n);
    hipFree(d_k);
    hipFree(p_A);
    hipFree(p_B);
    hipFree(p_C);
    kblasDestroy(&handle);
    magma_finalize();
    stop("Free memory");
  }

}
