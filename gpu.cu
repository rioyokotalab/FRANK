#include "hip/hip_runtime.h"
#include "functions.h"
#include "low_rank.h"
#include "timer.h"

#include <hipblas.h>
#include <magma_v2.h>
#include "kblas.h"
#include "testing_helper.h"
#include "batch_rand.h"
#include "batch_ara.h"

using namespace hicma;

int main(int argc, char** argv)
{
  int batchCount = 16;
  double tol = 1e-6;
  int block_size = 32;
  int ara_r = 10;
  std::vector<int> h_m;
  std::vector<int> h_n;
  std::vector<Dense> vecA;
  for (int b=0; b<batchCount; b++) {
    int N = 64+b;
    std::vector<double> randx(2*N);
    for (int i=0; i<2*N; i++) {
      randx[i] = drand48();
    }
    std::sort(randx.begin(), randx.end());
    Dense A(laplace1d, randx, N, N-2, 0, N);
    h_m.push_back(A.dim[0]);
    h_n.push_back(A.dim[1]);
    vecA.push_back(A);
  }
  int max_m = 0;
  int max_n = 0;
  for (int b=0; b<batchCount; b++) {
    max_m = std::max(max_m, h_m[b]);
    max_n = std::max(max_n, h_n[b]);
  }
  std::vector<double> h_A(max_m * max_n * batchCount);
  std::vector<double> h_U(max_m * max_n * batchCount);
  std::vector<double> h_V(max_n * max_n * batchCount);
  for (int b=0; b<batchCount; b++) {
    Dense A = vecA[b];
    for (int i=0; i<A.dim[0]; i++) {
      for (int j=0; j<A.dim[1]; j++) {
        h_A[i+j*A.dim[0]+b*max_m*max_n] = A(i,j);
      }
    }
  }
  kblasHandle_t handle;
  kblasRandState_t rand_state;
  kblasCreate(&handle);
  kblasInitRandState(handle, &rand_state, 16384*2, 0);
  kblasEnableMagma(handle);
  magma_init();
  int *d_m, *d_n, *d_k;
  hipMalloc( (void**)&d_m, batchCount * sizeof(int) );
  hipMalloc( (void**)&d_n, batchCount * sizeof(int) );
  hipMalloc( (void**)&d_k, batchCount * sizeof(int) );
  double *d_A, *d_U, *d_V;
  hipMalloc( (void**)&d_A, h_A.size() * sizeof(double) );
  hipMalloc( (void**)&d_U, h_U.size() * sizeof(double) );
  hipMalloc( (void**)&d_V, h_V.size() * sizeof(double) );
  double **p_A, **p_U, **p_V;
  hipMalloc( (void**)&p_A, batchCount * sizeof(double*) );
  hipMalloc( (void**)&p_U, batchCount * sizeof(double*) );
  hipMalloc( (void**)&p_V, batchCount * sizeof(double*) );
  generateDArrayOfPointers(d_A, p_A, max_m * max_n, batchCount, 0);
  generateDArrayOfPointers(d_U, p_U, max_m * max_n, batchCount, 0);
  generateDArrayOfPointers(d_V, p_V, max_n * max_n, batchCount, 0);
  kblas_ara_batch_wsquery<double>(handle, block_size, batchCount);
  kblasAllocateWorkspace(handle);
  hipMemcpy(d_m, &h_m[0], batchCount * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_n, &h_n[0], batchCount * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_A, &h_A[0], h_A.size() * sizeof(double), hipMemcpyHostToDevice);
  start("Batched RSVD");
  kblas_ara_batched(
                    handle, d_m, d_n, p_A, d_m, p_U, d_m, p_V, d_n, d_k,
                    tol, max_m, max_n, max_n, block_size, ara_r, rand_state, batchCount
                    );
  stop("Batched RSVD");
  std::vector<int> h_k(batchCount);
  hipMemcpy(&h_k[0], d_k, batchCount * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(&h_U[0], d_U, h_U.size() * sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(&h_V[0], d_V, h_V.size() * sizeof(double), hipMemcpyDeviceToHost);
  std::vector<LowRank> vecLR;
  for(int b=0; b<batchCount; b++) {
    LowRank LR(vecA[b].dim[0], vecA[b].dim[1], h_k[b]);
    Dense A = vecA[b];
    for (int i=0; i<LR.dim[0]; i++) {
      for (int j=0; j<LR.rank; j++) {
        LR.U(i,j) = h_U[i+j*LR.dim[0]+b*max_m*max_n];
      }
    }
    for (int i=0; i<LR.rank; i++) {
      for (int j=0; j<LR.dim[1]; j++) {
        LR.V(i,j) = h_V[i*LR.dim[1]+j+b*max_n*max_n];
      }
      LR.S(i,i) = 1;
    }
    vecLR.push_back(LR);
  }
  for (int b=0; b<batchCount; b++) {
    double diff = (vecA[b] - Dense(vecLR[b])).norm();
    double norm = vecA[b].norm();
    print("rank", h_k[b]);
    print("Rel. L2 Error", std::sqrt(diff/norm), false);
  }
  hipFree(p_A);
  hipFree(p_U);
  hipFree(p_V);
  hipFree(d_A);
  hipFree(d_U);
  hipFree(d_V);
  hipFree(d_m);
  hipFree(d_n);
  hipFree(d_k);
  return 0;
}
